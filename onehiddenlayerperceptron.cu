#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void kMartixByMatrixElementwise(const int nThreads, const float *m1, const float *m2, float *output) {
  /*  Computes the product of two arrays (elementwise multiplication).
    Inputs:
    m1: array
    m2: array
    output: array,the results of the multiplication are to be stored here
   */
  for (int i = blockIdx.x * blockDim.x + threadIdx.x;
      i < nThreads;
      i += blockDim.x * gridDim.x)
  {
    output[i] = m1[i] * m2[i];
  }
}
__device__ float* dMartixByMatrixElementwise(const float *m1, const float *m2, float *output, const int width, const int height){
  kMartixByMatrixElementwise <<< width, height >>> ( width * height, m1, m2, output );
  hipDeviceSynchronize();
  return output;
}
__global__ void kMartixSubstractMatrix(const int nThreads, const float *m1, const float *m2, float *output) {
  /*  Computes the (elementwise) difference between two arrays
    Inputs:
    m1: array
    m2: array
    output: array,the results of the computation are to be stored here
   */
  for (int i = blockIdx.x * blockDim.x + threadIdx.x;
      i < nThreads;
      i += blockDim.x * gridDim.x)
  {
    output[i] = m1[i] - m2[i];
  }
}
__device__ float* dMartixSubstractMatrix(const float *m1, const float *m2, float *output, const int width, const int height){
  kMartixSubstractMatrix <<< width, height >>> ( width * height, m1, m2, output );
  hipDeviceSynchronize();
  return output;
}
__global__ void kSigmoid(const int nThreads, float const *input, float *output){
  /*  Computes the value of the sigmoid function f(x) = 1/(1 + e^-x).
    Inputs:
    input: array
    output: array, the results of the computation are to be stored here
   */
  for (int i = blockIdx.x * blockDim.x + threadIdx.x;
      i < nThreads;
      i += blockDim.x * gridDim.x)
  {
    output[i] = 1.0 / (1.0 + std::exp(-input[i]));
  }
}
__device__ void dSigmoid(float const *input, float *output, const int height, const int width){
  kSigmoid <<< height, width >>> (height * width, input, output);
  hipDeviceSynchronize();
}
__global__ void kSigmoid_d(const int nThreads, float const *input, float *output) {
  /*  Computes the value of the sigmoid function derivative f'(x) = f(x)(1 - f(x)),
      where f(x) is sigmoid function.
    Inputs:
    input: array
    output: array, the results of the computation are to be stored here:
    x(1 - x) for every element of the input matrix m1.
   */
  for (int i = blockIdx.x * blockDim.x + threadIdx.x;
      i < nThreads;
      i += blockDim.x * gridDim.x)
  {
    output[i] = input[i] * (1 - input[i]);
  }
}
__device__ float* dSigmoid_d(float const *input, float *output, const int rows, const int columns){
  kSigmoid_d <<< rows, columns >>> (rows*columns, input, output);
  hipDeviceSynchronize();
  return output;
}

__global__ void kDot(const int nThreads, const float *m1, const float *m2, float *output, const int m1_rows , const int m1_columns, const int m2_columns ){
  /*  Computes the product of two matrices: m1 x m2.
    Inputs:
    m1: array, left matrix of size m1_rows x m1_columns
    m2: array, right matrix of size m1_columns x m2_columns (the number of rows in the right matrix
    must be equal to the number of the columns in the left one)
    output: array, the results of the computation are to be stored here:
    m1 * m2, product of two arrays m1 and m2, a matrix of size m1_rows x m2_columns
    m1_rows: int, number of rows in the left matrix m1
    m1_columns: int, number of columns in the left matrix m1
    m2_columns: int, number of columns in the right matrix m2
   */
  for (int i = blockIdx.x * blockDim.x + threadIdx.x;
      i < nThreads;
      i += blockDim.x * gridDim.x)
  {
    int r = (int)i / m2_columns;
    int c = i % m2_columns;
    float t_output = 0.f;
    for( int k = 0; k < m1_columns; ++k ) {
      t_output += m1[ r * m1_columns + k ] * m2[ k * m2_columns + c ];
    }
    output[i] = t_output;
  }
}

__device__ float* dDot(const float *m1, const float *m2, float *output, const int m1_rows , const int m1_columns, const int m2_columns ){
  kDot <<< m1_rows, m2_columns >>> (m1_rows * m2_columns, m1, m2, output, m1_rows , m1_columns, m2_columns );
  hipDeviceSynchronize();
  return output;
}

__global__ void kDot_m1_m2T(const int nThreads, const float *m1, const float *m2, float *output, const int m1_columns, const int m2_rows ){
  /*  Updates the output matrix with the product of two matrices: m1 and m2 transposed.
  Inputs:
  m1: array, left matrix of size m1_rows x m1_columns
  m2: array, right matrix of size m2_rows x m1_columns (m2 transposed will be of size m1_columns x m2_rows)
  output: array, the results of the computation are to be stored here:
  m1 * m2, product of two arrays m1 and m2, a matrix of size m1_rows x m2_rows
  m1_columns: int, number of columns in the left matrix m1
  m2_rows: int, number of rows in the left matrix m2
   */
  for (int i = blockIdx.x * blockDim.x + threadIdx.x;
      i < nThreads;
      i += blockDim.x * gridDim.x)
  {
    int r = (int)i / m2_rows;
    int c = i % m2_rows;
    float t_output = 0.0;
    int id_T;
    for( int k = 0; k < m1_columns; ++k ) {
      id_T = c * m1_columns + k;
      t_output += m1[ r * m1_columns + k ] * m2[ id_T ];
    }
    output[i] = t_output;
  }
}

__device__ float* dDot_m1_m2T(const float *m1, const float *m2, float *output, const int m1_rows , const int m1_columns, const int m2_rows )
{
  kDot_m1_m2T <<< m1_rows, m2_rows >>> ( m1_rows * m2_rows, m1, m2, output, m1_columns, m2_rows );
  hipDeviceSynchronize();
  return output;
}

__global__ void kDot_m1T_m2(const int nThreads, const float *m1, const float *m2, float *output, const int m1_rows,
    const int m1_columns, const int m2_columns ){
  /*  Increments the output matrix with the product of two matrices: m1 transposed and m2.
    Inputs:
    m1: array, left matrix of size m1_rows x m1_columns (m1 transposed will be of size m1_columns x m1_rows)
    m2: array, right matrix of size m1_rows x m2_columns
    output: array, the results of the computation are to be stored here:
    m1 * m2, product of two arrays m1 and m2, a matrix of size m1_columns x m2_columns
    m1_rows: int, number of rows in the left matrix m1
    m1_columns: int, number of columns in the left matrix m1
    m2_rows: int, number of rows in the left matrix m2
   */
  for (int i = blockIdx.x * blockDim.x + threadIdx.x;
      i < nThreads;
      i += blockDim.x * gridDim.x)
  {
    int r = (int)i / m2_columns;
    int c = i % m2_columns;
    int id_T;
    float t_output = 0.0;
    for( int k = 0; k < m1_rows; ++k ) {
      id_T = k * m1_columns + r;
      t_output += m1[ id_T ] * m2[ k * m2_columns + c ];
    }
    output[i] += t_output;
  }
}
__device__ void dDot_m1T_m2(const float *m1, const float *m2, float *output, const int m1_height , const int m1_width, const int m2_width )
{
  kDot_m1T_m2 <<< m1_width, m2_width >>> (m1_width * m2_width, m1, m2, output, m1_height, m1_width, m2_width );
  hipDeviceSynchronize();
}
__device__ void kPrintMatrix (const float* M, int h, int w) {
  /*  Prints out the input array as h x w matrix.
    Inputs:
    m: vector, matrix of size n_rows x n_columns
    h: int, number of rows in the matrix M
    w: int, number of columns in the matrix M
   */
  for (int i = 0; i < h; i++){
    for (int j = 0; j < w; j++){
      printf("%f  ", M[i*w+j]);
    }
    printf("\n");
  }
  printf("\n");
}
__global__ void kFit(	const float* X, const int X_w, const int X_h,
    const float* y, const int y_w,
    float* l1, const int l1_w, float* l_1_d,
    float* pred, float* pred_d,
    float* W0,
    float* W1,
    float* buffer
    )
{
  for (unsigned i = 0; i < 50; ++i) {
    dSigmoid(dDot(X, W0, l1, X_h, X_w, l1_w), l1, X_h, l1_w);
    dSigmoid(dDot(l1, W1, pred, X_h, l1_w, y_w), pred, X_h, y_w);
    dMartixByMatrixElementwise(dMartixSubstractMatrix(y, pred, pred_d, X_h, y_w), dSigmoid_d(pred, buffer, X_h, y_w), pred_d, X_h, y_w );
    dMartixByMatrixElementwise(dDot_m1_m2T(pred_d, W1, l_1_d, X_h, y_w, l1_w), dSigmoid_d(l1, buffer, X_h, l1_w), l_1_d, X_h, l1_w);
    dDot_m1T_m2( l1, pred_d, W1, X_h, l1_w, y_w );
    dDot_m1T_m2( X, l_1_d, W0, X_h, X_w, l1_w );
  }
}


int main(void){
  const int TRAINING_SIZE = 4;
  const int TRAINING_DIM = 4;
  const int L1_SIZE = 8;
  // X, the first 4 lines from Iris dataset
  float h_X[TRAINING_SIZE*TRAINING_DIM] = {	5.1, 3.5, 1.4, 0.2,
    4.9, 3.0, 1.4, 0.2,
    6.2, 3.4, 5.4, 2.3,
    5.9, 3.0, 5.1, 1.8 };
  const signed int X_size = sizeof(h_X);
  float *d_X;
  hipMalloc(&d_X, X_size);
  hipMemcpy(d_X, h_X, X_size, hipMemcpyHostToDevice);
  //WEIGHTS_0
  const long signed int W0_size = L1_SIZE*TRAINING_DIM*sizeof(float);
  float *h_W0 = (float*)malloc(W0_size);
  for (int i = 0; i < L1_SIZE*TRAINING_DIM; i++){
    h_W0[i] = 0.1 * (2.0*rand()/RAND_MAX-1.0);
  }
  float *d_W0;
  hipMalloc(&d_W0, W0_size);
  hipMemcpy(d_W0, h_W0, W0_size, hipMemcpyHostToDevice);
  //LAYER_1, LAYER_1_DELTA AND BUFFER OF LAYER 1 SIZE
  const long signed int L1_size = L1_SIZE*TRAINING_SIZE*sizeof(float);
  float* h_layer_1 = (float*)malloc(L1_size);
  float* h_layer_1_delta = (float*)malloc(L1_size);
  float* h_buffer = (float*)malloc(L1_size);
  for (int i = 0; i < L1_SIZE*TRAINING_SIZE; i++){
    h_layer_1[i] = 0.0;
    h_buffer[i] = 0.0;
    h_layer_1_delta[i] = 0.0;
  }
  float *d_layer_1;
  hipMalloc(&d_layer_1, L1_size);
  hipMemcpy(d_layer_1, h_layer_1, L1_size, hipMemcpyHostToDevice);
  float *d_buffer;
  hipMalloc(&d_buffer, L1_size);
  hipMemcpy(d_buffer, h_buffer, L1_size, hipMemcpyHostToDevice);
  float *d_layer_1_delta;
  hipMalloc(&d_layer_1_delta, L1_size);
  hipMemcpy(d_layer_1_delta, h_layer_1_delta, L1_size, hipMemcpyHostToDevice);
  //WEIGHTS_1
  const long signed int W1_size = L1_SIZE*sizeof(float);
  float *h_W1 = (float*)malloc(W1_size);
  for (int i = 0; i < L1_SIZE; i++){
    h_W1[i] = 0.1* (2.0*rand()/RAND_MAX-1.0);
  }
  float *d_W1;
  hipMalloc(&d_W1, W1_size);
  hipMemcpy(d_W1, h_W1, W1_size, hipMemcpyHostToDevice);
  //Y
  float h_y[4] = {	0,
    0,
    1,
    1 };
  const signed int y_size = sizeof(h_y);
  float *d_y;
  hipMalloc(&d_y, y_size);
  hipMemcpy(d_y, h_y, y_size, hipMemcpyHostToDevice);
  //PRED AND PRED_DELTA
  float* h_pred = (float*)malloc(y_size);
  float* h_pred_delta = (float*)malloc(y_size);
  for (int i = 0; i < TRAINING_SIZE; i++){
    h_pred[i] = 0.0;
    h_pred_delta[i] = 0.0;
  }
  float *d_pred;
  hipMalloc(&d_pred, y_size);
  hipMemcpy(d_pred, h_pred, y_size, hipMemcpyHostToDevice);
  float *d_pred_delta;
  hipMalloc(&d_pred_delta, y_size);
  hipMemcpy(d_pred_delta, h_pred_delta, y_size, hipMemcpyHostToDevice);

  
    kFit <<< 1, 1 >>> (	d_X, TRAINING_DIM, TRAINING_SIZE,
        d_y, 1,
        d_layer_1, L1_SIZE, d_layer_1_delta,
        d_pred,
        d_pred_delta,
        d_W0,
        d_W1,
        d_buffer);
    hipMemcpy(h_pred, d_pred, y_size, hipMemcpyDeviceToHost);
    hipFree(d_pred);
    hipFree(d_X);
    hipFree(d_y);
    hipFree(d_layer_1_delta);
    hipFree(d_pred_delta);
    hipFree(d_W0);
    hipFree(d_W1);
    hipFree(d_buffer);

    free(h_layer_1_delta);
    free(h_pred_delta);
    free(h_W0);
    free(h_W1);
    free(h_buffer);

    for (int i = 0; i < TRAINING_SIZE; i++){
      printf("Prediction[%i] : %f True Value[%i] : %f Error[%i] : %f\n", i, h_pred[i], i, h_y[i], i, h_pred[i] - h_y[i]);
    }
    free(h_pred);
}
