#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <vector>
#include <string>
#include <stdio.h>
#include "npy.hpp"


__global__ void kMartixByMatrixElementwise(const int nThreads, const float *m1, const float *m2, float *output) {
  /*  Computes the product of two arrays (elementwise multiplication).
    Inputs:
    m1: array
    m2: array
    output: array,the results of the multiplication are to be stored here
   */
  for (int i = blockIdx.x * blockDim.x + threadIdx.x;
      i < nThreads;
      i += blockDim.x * gridDim.x)
  {
    output[i] = m1[i] * m2[i];
  }
}
__device__ float* dMartixByMatrixElementwise(const float *m1, const float *m2, float *output, const int width, const int height){
  kMartixByMatrixElementwise <<< width, height >>> ( width * height, m1, m2, output );
  hipDeviceSynchronize();
  return output;
}
__global__ void kMartixSubstractMatrix(const int nThreads, const float *m1, const float *m2, float *output) {
  /*  Computes the (elementwise) difference between two arrays
    Inputs:
    m1: array
    m2: array
    output: array,the results of the computation are to be stored here
   */
  for (int i = blockIdx.x * blockDim.x + threadIdx.x;
      i < nThreads;
      i += blockDim.x * gridDim.x)
  {
    output[i] = m1[i] - m2[i];
  }
}
__device__ float* dMartixSubstractMatrix(const float *m1, const float *m2, float *output, const int width, const int height){
  kMartixSubstractMatrix <<< width, height >>> ( width * height, m1, m2, output );
  hipDeviceSynchronize();
  return output;
}
__global__ void kSigmoid(const int nThreads, float const *input, float *output){
  /*  Computes the value of the sigmoid function f(x) = 1/(1 + e^-x).
    Inputs:
    input: array
    output: array, the results of the computation are to be stored here
   */
  for (int i = blockIdx.x * blockDim.x + threadIdx.x;
      i < nThreads;
      i += blockDim.x * gridDim.x)
  {
    output[i] = 1.0 / (1.0 + std::exp(-input[i]));
  }
}
__device__ void dSigmoid(float const *input, float *output, const int height, const int width){
  kSigmoid <<< height, width >>> (height * width, input, output);
  hipDeviceSynchronize();
}
__global__ void kSigmoid_d(const int nThreads, float const *input, float *output) {
  /*  Computes the value of the sigmoid function derivative f'(x) = f(x)(1 - f(x)),
      where f(x) is sigmoid function.
    Inputs:
    input: array
    output: array, the results of the computation are to be stored here:
    x(1 - x) for every element of the input matrix m1.
   */
  for (int i = blockIdx.x * blockDim.x + threadIdx.x;
      i < nThreads;
      i += blockDim.x * gridDim.x)
  {
    output[i] = input[i] * (1 - input[i]);
  }
}
__device__ float* dSigmoid_d(float const *input, float *output, const int rows, const int columns){
  kSigmoid_d <<< rows, columns >>> (rows*columns, input, output);
  hipDeviceSynchronize();
  return output;
}

__global__ void kDot(const int nThreads, const float *m1, const float *m2, float *output, const int m1_rows , const int m1_columns, const int m2_columns ){
  /*  Computes the product of two matrices: m1 x m2.
    Inputs:
    m1: array, left matrix of size m1_rows x m1_columns
    m2: array, right matrix of size m1_columns x m2_columns (the number of rows in the right matrix
    must be equal to the number of the columns in the left one)
    output: array, the results of the computation are to be stored here:
    m1 * m2, product of two arrays m1 and m2, a matrix of size m1_rows x m2_columns
    m1_rows: int, number of rows in the left matrix m1
    m1_columns: int, number of columns in the left matrix m1
    m2_columns: int, number of columns in the right matrix m2
   */
  for (int i = blockIdx.x * blockDim.x + threadIdx.x;
      i < nThreads;
      i += blockDim.x * gridDim.x)
  {
    int r = (int)i / m2_columns;
    int c = i % m2_columns;
    float t_output = 0.f;
    for( int k = 0; k < m1_columns; ++k ) {
      t_output += m1[ r * m1_columns + k ] * m2[ k * m2_columns + c ];
    }
    output[i] = t_output;
  }
}

__device__ float* dDot(const float *m1, const float *m2, float *output, const int m1_rows , const int m1_columns, const int m2_columns ){
  kDot <<< m1_rows, m2_columns >>> (m1_rows * m2_columns, m1, m2, output, m1_rows , m1_columns, m2_columns );
  hipDeviceSynchronize();
  return output;
}

__global__ void kDot_m1_m2T(const int nThreads, const float *m1, const float *m2, float *output, const int m1_columns, const int m2_rows ){
  /*  Updates the output matrix with the product of two matrices: m1 and m2 transposed.
  Inputs:
  m1: array, left matrix of size m1_rows x m1_columns
  m2: array, right matrix of size m2_rows x m1_columns (m2 transposed will be of size m1_columns x m2_rows)
  output: array, the results of the computation are to be stored here:
  m1 * m2, product of two arrays m1 and m2, a matrix of size m1_rows x m2_rows
  m1_columns: int, number of columns in the left matrix m1
  m2_rows: int, number of rows in the left matrix m2
   */
  for (int i = blockIdx.x * blockDim.x + threadIdx.x;
      i < nThreads;
      i += blockDim.x * gridDim.x)
  {
    int r = (int)i / m2_rows;
    int c = i % m2_rows;
    float t_output = 0.0;
    int id_T;
    for( int k = 0; k < m1_columns; ++k ) {
      id_T = c * m1_columns + k;
      t_output += m1[ r * m1_columns + k ] * m2[ id_T ];
    }
    output[i] = t_output;
  }
}

__device__ float* dDot_m1_m2T(const float *m1, const float *m2, float *output, const int m1_rows , const int m1_columns, const int m2_rows )
{
  kDot_m1_m2T <<< m1_rows, m2_rows >>> ( m1_rows * m2_rows, m1, m2, output, m1_columns, m2_rows );
  hipDeviceSynchronize();
  return output;
}

__global__ void kDot_m1T_m2(const int nThreads, const float *m1, const float *m2, float *output, const int m1_rows,
    const int m1_columns, const int m2_columns ){
  /*  Increments the output matrix with the product of two matrices: m1 transposed and m2.
    Inputs:
    m1: array, left matrix of size m1_rows x m1_columns (m1 transposed will be of size m1_columns x m1_rows)
    m2: array, right matrix of size m1_rows x m2_columns
    output: array, the results of the computation are to be stored here:
    m1 * m2, product of two arrays m1 and m2, a matrix of size m1_columns x m2_columns
    m1_rows: int, number of rows in the left matrix m1
    m1_columns: int, number of columns in the left matrix m1
    m2_rows: int, number of rows in the left matrix m2
   */
  for (int i = blockIdx.x * blockDim.x + threadIdx.x;
      i < nThreads;
      i += blockDim.x * gridDim.x)
  {
    int r = (int)i / m2_columns;
    int c = i % m2_columns;
    int id_T;
    float t_output = 0.0;
    for( int k = 0; k < m1_rows; ++k ) {
      id_T = k * m1_columns + r;
      t_output += m1[ id_T ] * m2[ k * m2_columns + c ];
    }
    output[i] += t_output;
  }
}
__device__ void dDot_m1T_m2(const float *m1, const float *m2, float *output, const int m1_height , const int m1_width, const int m2_width )
{
  kDot_m1T_m2 <<< m1_width, m2_width >>> (m1_width * m2_width, m1, m2, output, m1_height, m1_width, m2_width );
  hipDeviceSynchronize();
}
__device__ void kPrintMatrix (const float* M, int h, int w) {
  /*  Prints out the input array as h x w matrix.
    Inputs:
    m: vector, matrix of size n_rows x n_columns
    h: int, number of rows in the matrix M
    w: int, number of columns in the matrix M
   */
  for (int i = 0; i < h; i++){
    for (int j = 0; j < w; j++){
      printf("%f  ", M[i*w+j]);
    }
    printf("\n");
  }
  printf("\n");
}

__global__ void kForwardPass(	const float* X, const int X_w, const int X_h,
    const int y_w,
    float* l1, const int l1_w,
    float* pred,
    float* W0,
    float* W1
    )
    { 
      
      dSigmoid(dDot(X, W0, l1, X_h, X_w, l1_w), l1, X_h, l1_w);
      
      dSigmoid(dDot(l1, W1, pred, X_h, l1_w, y_w), pred, X_h, y_w);
  
      dDot(l1, W1, pred, X_h, l1_w, y_w);
  
      
    }


__global__ void kFit(	const float* X, const int X_w, const int X_h,
    const float* y, const int y_w,
    float* l1, const int l1_w, float* l_1_d,
    float* pred, float* pred_d,
    float* W0,
    float* W1,
    float* buffer
    )
{
  for (unsigned i = 0; i < 50; ++i) {
    //Forward pass
    dSigmoid(dDot(X, W0, l1, X_h, X_w, l1_w), l1, X_h, l1_w);
    dSigmoid(dDot(l1, W1, pred, X_h, l1_w, y_w), pred, X_h, y_w);

    //Loss calculation
    dMartixByMatrixElementwise(dMartixSubstractMatrix(y, pred, pred_d, X_h, y_w), dSigmoid_d(pred, buffer, X_h, y_w), pred_d, X_h, y_w );
    dMartixByMatrixElementwise(dDot_m1_m2T(pred_d, W1, l_1_d, X_h, y_w, l1_w), dSigmoid_d(l1, buffer, X_h, l1_w), l_1_d, X_h, l1_w);
    
    //Backward 
    dDot_m1T_m2( l1, pred_d, W1, X_h, l1_w, y_w );
    dDot_m1T_m2( X, l_1_d, W0, X_h, X_w, l1_w );
  }
}

std::vector<char> get_the_bytes(std::string filename) {
    std::ifstream input(filename, std::ios::binary);
    std::vector<char> bytes(
        (std::istreambuf_iterator<char>(input)),
        (std::istreambuf_iterator<char>()));

    input.close();
    return bytes;
}


// Model to replicate
// class LinearNetwork(torch.nn.Module):
//     def __init__(self, input_channels, num_classes):
//         super(LinearNetwork, self).__init__()
//         self.fully_connected_network1 = torch.nn.Linear(
//             in_features=input_channels, out_features=100
//         )
//         self.fully_connected_network2 = torch.nn.Linear(
//             in_features=100, out_features=num_classes
//         )

//     def forward(self, x):
//         x = x.view(x.shape[0], -1)
//         x = torch.sigmoid(self.fully_connected_network1(x))
//         x = self.fully_connected_network2(x)

//         return x

// # Initialize network
// model = LinearNetwork(input_channels = 122880, num_classes = 2).to(device=device)




int main(void){

  std::vector<unsigned long> input_shape {1,122880};

  // input x weights1 + bias
  std::vector<unsigned long> fullyconnected0weight_shape {100,122880};
  std::vector<unsigned long> fullyconnected0bias_shape {100};

  // sigmoid(output) x  weights2 + bias
  std::vector<unsigned long> fullyconnected1weight_shape {2,100};
  std::vector<unsigned long> fullyconnected1bias_shape {2};
  
  bool fortran_order = false;

  std::vector<float> input_data;

  std::vector<float> fullyconnected0weight_data;
  std::vector<float> fullyconnected0bias_data;
  
  std::vector<float> fullyconnected1weight_data;
  std::vector<float> fullyconnected1bias_data;
  
  const std::string input_path {"./sample_input/0/features_1001423e-20220630222223.655.npy"} ;
  // const std::string input_path {"./sample_input/1/features_1001423e-20220628162912.729.npy"} ;

  const std::string fullyconnected0bias_path {"./weights_classification/fully_connected_network1.bias.npy"};
  const std::string fullyconnected0weight_path {"./weights_classification/fully_connected_network1.weight.npy"};
  
  const std::string fullyconnected1bias_path {"./weights_classification/fully_connected_network2.bias.npy"};
  const std::string fullyconnected1weight_path {"./weights_classification/fully_connected_network2.weight.npy"};
  
  std::cout << "Loading tensors" << std::endl;

  npy::LoadArrayFromNumpy(input_path, input_shape, fortran_order, input_data);

  npy::LoadArrayFromNumpy(fullyconnected0weight_path, fullyconnected0weight_shape, fortran_order, fullyconnected0weight_data);
  npy::LoadArrayFromNumpy(fullyconnected0bias_path, fullyconnected0bias_shape, fortran_order, fullyconnected0bias_data);
  
  npy::LoadArrayFromNumpy(fullyconnected1weight_path, fullyconnected1weight_shape, fortran_order, fullyconnected1weight_data);
  npy::LoadArrayFromNumpy(fullyconnected1bias_path, fullyconnected1bias_shape, fortran_order, fullyconnected1bias_data);

  std::cout << "Input shape : " << input_data.size() << std::endl;

  std::cout << "Weights 1 shape : " << fullyconnected0weight_data.size() << std::endl;
  std::cout << "Bias 1 shape : " << fullyconnected0bias_data.size() << std::endl;
  
  std::cout <<  "Weights 2 shape : " << fullyconnected1weight_data.size()  << std::endl;
  std::cout <<  "Bias 2 shape : " << fullyconnected1bias_data.size() << std::endl;
  
  const signed int X_size = 122880;
  float *h_X = &input_data[0];

  float *d_X;
  hipMalloc(&d_X, X_size);
  hipMemcpy(d_X, h_X, X_size, hipMemcpyHostToDevice);
  
  //LAYER_1, LAYER_1_DELTA AND BUFFER OF LAYER 1 SIZE
  const long signed int L1_size = 122880*1*sizeof(float);

  float* h_layer_1 = (float*)malloc(L1_size);
  float* h_layer_1_delta = (float*)malloc(L1_size);
  float* h_buffer = (float*)malloc(L1_size);
  for (int i = 0; i < 122880*1; i++){
        h_layer_1[i] = 0.0;
        h_buffer[i] = 0.0;
        h_layer_1_delta[i] = 0.0;
  }

  float *d_layer_1;
  hipMalloc(&d_layer_1, L1_size);
  hipMemcpy(d_layer_1, h_layer_1, L1_size, hipMemcpyHostToDevice);
  
  //WEIGHTS_0
  const long signed int W0_size = 122880*100*sizeof(float);
  float *h_W0 = &fullyconnected0weight_data[0];
  float *d_W0;
  hipMalloc(&d_W0, W0_size);
  hipMemcpy(d_W0, h_W0, W0_size, hipMemcpyHostToDevice);

  //WEIGHTS_1
  const long signed int W1_size = 100*2*sizeof(float);
  float *h_W1 = &fullyconnected1weight_data[0];
  float *d_W1;
  hipMalloc(&d_W1, W1_size);
  hipMemcpy(d_W1, h_W1, W1_size, hipMemcpyHostToDevice);

  
  float *d_pred;
  float* h_pred = (float*)malloc(2);

  hipMalloc(&d_pred, 2); 
  hipMemcpy(d_pred, h_pred, 2, hipMemcpyHostToDevice);


  kForwardPass <<< 1, 1 >>> (	d_X, 122880, 1,
						2, 
						d_layer_1, L1_size,
						d_pred,
						d_W0,
						d_W1);

  
  hipMemcpy(h_pred, d_pred, 2, hipMemcpyDeviceToHost);

  hipFree(d_pred);
  hipFree(d_X);
  hipFree(d_W0);
  hipFree(d_W1);
  std::cout << "Prediction : "<< h_pred[0] << " " << h_pred[1] << std::endl;
  
}
